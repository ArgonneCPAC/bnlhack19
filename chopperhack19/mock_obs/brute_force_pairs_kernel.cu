
#include <hip/hip_runtime.h>
extern "C"{

__global__ void brute_force_pairs_kernel(
    float* x1, float* y1, float* z1, float* w1,
    float* x2, float* y2, float* z2, float* w2,
    float* rbins_squared, float* result,
    int n1, int n2, int nbins) {
    // array attributes must be explicitly passed in.
    /*
       Direct translation of the Numba "double_chop_pairs_cuda" kernel.
    */

    size_t start = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x;

//    if (start == 0)
//        printf("%i, %i\n", n1, nbins);

    for (size_t i = start; i < n1; i += stride) {
        float px = x1[i];
        float py = y1[i];
        float pz = z1[i];
        float pw = w1[i];

        for (size_t j = 0; j < n2; j++) {
            float qx = x2[j];
            float qy = y2[j];
            float qz = z2[j];
            float qw = w2[j];

            float dx = px - qx;
            float dy = py - qy;
            float dz = pz - qz;
            float wprod = pw * qw;
            float dsq = dx * dx + dy * dy + dz * dz;

            size_t k = nbins - 1;
            while (dsq <= rbins_squared[k]) {
                atomicAdd(&(result[k-1]), wprod);
                k -= 1;
                if (k <= 0) break;
            }
        }
    }
}

}
